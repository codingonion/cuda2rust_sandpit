//#include <iostream>
#include<stdio.h>
#include <math.h>
#include <stdlib.h>
//#include <cuda.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define W 512
#define H (512 * 1024)

// input is a host device mem vector
extern "C" {
void cuda_call_spaxy(const float *x, float *y, int N, float alpha ) {
    hipError_t cudaStat;
    // create a device handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // allocate the GPU memory for x, and y
    float *d_x, *d_y;

    cudaStat = hipMalloc ((void**)&d_x, N*sizeof(float));
    cudaStat = hipMalloc ((void**)&d_y, N*sizeof(float));

    float *alpha_;
    alpha_ = &alpha;

    hipblasSetVector(N, sizeof(x[0]), x, 1, d_x, 1);
    hipblasSetVector(N, sizeof(y[0]), y, 1, d_y, 1);

    hipblasSaxpy(handle, N, alpha_, d_x, 1, d_y, 1);

    hipblasGetVector(N, sizeof(d_y[0]), d_y, 1, y, 1);

    hipFree(d_x); hipFree(d_y);

    // destroy the hanle
    hipblasDestroy(handle);
}
}


__global__ void expand_kernel(
    const float* vector, 
    const unsigned vlen, 
    float* matrix,
    const unsigned mdim
    ) {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        float myval = vector[idx%vlen];
        while (idx < mdim*vlen){
            matrix[idx] = myval;
            idx += gridDim.x*blockDim.x;
        }
    }



extern "C" {
void call_reshape(const float *x, float *y, int w, int num_actions){

    float *d_x, *d_y;
    hipMalloc ((void**)&d_x, w*sizeof(float));
    hipMalloc ((void**)&d_y, w*num_actions*sizeof(float));
    hipMemcpy(d_x, x, w * ( sizeof(float)), hipMemcpyHostToDevice);

    expand_kernel<<<w, 256>>>(d_x, w, d_y, num_actions);

    hipMemcpy(y, d_y, w * num_actions * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_x); hipFree(d_y);
}
}


