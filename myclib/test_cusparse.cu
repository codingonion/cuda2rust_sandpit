#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
#include <hipsparse.h>

// What we are interested in constructing a CSR matrix
// Assume that we already have a sparse matrix on host memory using 
// CXSparse and we just want to convert it to device memory 
extern "C" {
void csr_spmv(
    int *csr_row, 
    int *csr_col, 
    float *csr_vals, 
    float *x,
    float *y,
    int nnz, 
    int sizeof_row, 
    int m, 
    int n
    ) {
    // sizeof_row is the size of csr_row
    // sizeof_col is the size of csr_col
    // m number of rows in the matrix
    // n number of cols in the matrix
    // nnz is the size of the csr_vals
    // create a sparse handle
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);
    hipsparseSpMatDescr_t descrC = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    int *dCsrRowPtr, *dCsrColPtr;
    float *dCsrValPtr;

    // allocate device memory to store the sparse CSR
    hipMalloc((void **)&dCsrValPtr, sizeof(float) * nnz);
    hipMalloc((void **)&dCsrColPtr, sizeof(int) * nnz);
    hipMalloc((void **)&dCsrRowPtr, sizeof(int) * sizeof_row);

    // Free the device memory allocated to the coo ptrs once they
    // the conversion from coo to csr has been completed
    hipMemcpy(dCsrValPtr, csr_vals, sizeof(float) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dCsrColPtr, csr_col, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dCsrRowPtr, csr_row, sizeof(int) * sizeof_row, hipMemcpyHostToDevice);

    // create the sparse CSR matrix in device memory
    status = hipsparseCreateCsr(
        &descrC, // MATRIX DESCRIPTION
        m, // NUMBER OF ROWS
        n, // NUMBER OF COLS
        nnz, // NUMBER OF NON ZERO VALUES
        dCsrRowPtr, // ROWS OFFSETS
        dCsrColPtr, // COL INDICES
        dCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    );

    float alpha = 1.0;
    float beta = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecX, vecY;
    float *dX, *dY;
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    hipMalloc((void**)&dX, m * sizeof(float));
    hipMalloc((void**)&dY, n * sizeof(float));

    // copy the vector from host memory to device memory
    hipMemcpy(dX, x, m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dY, y, n * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    hipsparseCreateDnVec(&vecX, m, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, n, dY, HIP_R_32F);

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrC, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alpha, descrC, vecX, &beta, vecY, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

    // Any algorithms get inserted here

    hipMemcpy(y, dY, n *sizeof(float), hipMemcpyDeviceToHost);

    //destroy the vector descriptors
    hipsparseDestroySpMat(descrC);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);

    // Free the device memory
    hipFree(dCsrColPtr);
    hipFree(dCsrRowPtr);
    hipFree(dCsrValPtr);
    hipFree(dX);
    hipFree(dY);
    hipFree(dBuffer);

}

}